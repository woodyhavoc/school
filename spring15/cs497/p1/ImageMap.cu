#include "hip/hip_runtime.h"
/**
* Author: Aaron Woods
* Class: CS497
* Assignment: P1
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

/**
* This device function accepts an integer array filled with data from an
* image file.  Each index of the array is looped through, and if the
* value at the current index is equal to the index variable, it is set to 1,
* otherwise it is set to 0. This creates a binary array, where the index value
* at the indicies containig a one are the integer values of the various
* colors in the image file.
*
* @param input - The array to be converted into a binary array.
* @param outut - The array in which the binary information will be stored.
* @param maxCount - The size of the input array.
* @param stepSize - The amount by which each thread is incremented on
* 					each iteration of the loop.
*/
__global__ void fixInput(int* input, int* output, int maxCount, int stepSize);

/**
* This device function accepts an integer array which will have been converted
* to a binary array in the previous device function.  A prefix sum is performed
* on the array.  At the end of the prefix sum process, the sum of all the values
* in the array will be stored in index 0.
* 
* @param input - The array, which contains only 0's and 1's.
* @param maxCount - The size of the input array.
* @param stepSize - The amount by which each thread is incrememnted on
* 					each iteration of the loop. 
*/
__global__ void prefixSum(int* input, int maxCount, int stepSize);

/**
* This device function accepts an integer array which contains the prefix sum
* information generated in the prefixSum() function.  The array is iterated 
* through and each time that a value at a particular index is not equal to the value 
* of the index which came before it, the value maxCount - the index is stored in
* the output array at the index represented by the value at the current index of
* the input array.  A more clear depiction is given below:
*
* 			output[input[tid]] = maxCount - tid;
* 
* @param input - The array which contains the prefix sum information.
* @param output - The array which the unique color information will be
*				  stored in.
* @param maxCount - The size of the input array.
* @param stepSize - The amount by which each thread is incremented on each
*					each iteration of the loop.
*/
__global__ void fillArray(int* input, int* output, int maxCount, int stepSize);

/**
* Fills the output array sequentially with the image color region
* information.
* 
* @param input - The array that contains the binary image color region
* 				 information.
* @param output - The array that will hold the actual integer values of the
*				  various color regions.
* @param - arraySize - The size of the output array. 
*/
void seqFill(int* input, int* output, int arraySize);

/**
* Prints the results of the image color mapping.  The number of regions
* is first printed, followed by the 50 regions with the smallest value.
*
* @param input - The array that contains the image color mapping information.
* @param arraySize - The size of the input array.
*/
void printResults(int* input, int arraySize);

// Block and thread dimensions
const dim3 NUM_BLOCKS(1);
const dim3 NUM_THREADS(256);

/**
* Main function, calls the appropriate functions to evaluate the image
* color region mapping.
* 
* @param argc - The number of command-line arguments including the program name.
* @param argv - The string value of each command-line argument including
*				the program name.
*/
int main(int argc, char** argv)
{
	if(argc != 2)
	{
		fprintf(stderr, "Incorrect number of arguments.\nUsage: <program name> <image file>\n");
		exit(1);
	}

	int* devColorArray, *devFileData, *devFillArray;
	int* colorArray = NULL;
	int* fileData = NULL;
	int* fill = NULL;
		
	FILE* pFile = fopen(argv[1], "rb");
	
	int sizeInt = sizeof(int);
	int stepSize = NUM_BLOCKS.x * NUM_THREADS.x;
	int height, width, sizeFile, maxCount, numRegions, i;	
	
	if(pFile == NULL)
	{
		fprintf(stderr, "File error.\n");
		exit(1);
	}

	fread(&height, sizeInt, 1, pFile);
	fread(&width, sizeInt, 1, pFile);

	sizeFile = width * height * sizeInt;
	maxCount = width * height;
	
	fileData = (int*)malloc(sizeFile + sizeInt);
	
	fread(fileData, sizeInt, width*height, pFile);
	
	hipMalloc((void**)&devColorArray, sizeFile);
	hipMalloc((void**)&devFileData, sizeFile);
		
	hipMemcpy(devFileData, fileData, sizeFile, hipMemcpyHostToDevice);
	
	fixInput<<<NUM_BLOCKS, NUM_THREADS>>>(devFileData, devColorArray, maxCount, stepSize);
	prefixSum<<<NUM_BLOCKS, NUM_THREADS>>>(devFileData, maxCount, stepSize);

	hipMemcpy(fileData, devFileData, sizeFile, hipMemcpyDeviceToHost);
	hipMemcpy(colorArray, devColorArray, sizeFile, hipMemcpyDeviceToHost);

	numRegions = fileData[0];

	fill = (int*)malloc(numRegions * sizeInt);
	hipMalloc((void**)&devFillArray, numRegions * sizeInt);
	hipMemset(devFillArray, 0, numRegions * sizeInt);

	/*
	* The multi-threaded portion to fill the output
	* array would not work, so that is why there is also
	* a sequential function that populates the output array.
	* I left the multi-threaded function so you could see
	* what I was attempting to do.
	*/

	//fillArray<<<NUM_BLOCKS, NUM_THREADS>>>(devFileData, devFillArray, maxCount, stepSize);
	
	//hipMemcpy(fill, devFillArray, numRegions * sizeInt, hipMemcpyDeviceToHost);

	seqFill(colorArray, fill, maxCount, numRegions);

	printResults(fill, numRegions);

	return 0;
}

__global__ void fixInput(int* input, int* output, int maxCount, int stepSize)
{
	int tid;

	for(tid = blockIdx.x * blockDim.x + threadIdx.x; tid < maxCount; tid += stepSize)
	{
		input[tid] = input[tid] == tid ? 1 : 0;
		output[tid] = input[tid];
	}

	__syncthreads();
}

__global__ void prefixSum(int* input, int maxCount, int stepSize)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 1;
	int tid = threadIndex;
	
	while(i < maxCount)
	{
		while(tid + i < maxCount)
		{
			input[tid] += input[tid + i];
			tid += stepSize;		
		}

		__syncthreads();
		i <<= 1;
		tid = threadIndex;
	}

	__syncthreads();	
}

__global__ void fillArray(int* input, int* output, int maxCount, int stepSize)
{
	int tid;

	for(tid = maxCount - (blockIdx.x * blockDim.x + threadIdx.x) - 1; tid >= 0; tid -= stepSize)
	{
		if(input[tid] != input[tid-1])
		{
			output[input[tid]] = maxCount - tid;
		}
	}

	__syncthreads();
}

void seqFill(int* input, int* output, int maxCount, int arraySize)
{
	int i, index = 0;

	for(i = 0; i < maxCount; i++)
	{
		if(input[i] == 1)
		{
			output[index++] = i;
		}
	}

	if(index != arraySize)
	{
		fprintf(stderr, "Actual output array size: %d != expected output array size: %d\n",
		index, arraySize);

		exit(1);
	}
}

void printResults(int* input, int arraySize)
{
	printf("\n\nNumber of regions in image: %d\n", arraySize);
	printf("The first 50 regions are:\n");

	int i;

	for(i = 0; i < 50; i++)
	{
		printf("%d\t", input[i]);

		if((i+1) % 5 == 0)
		{
			printf("\n");
		}
	}
}















